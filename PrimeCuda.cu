#include "hip/hip_runtime.h"
#include "PrimeCuda.cuh"

typedef unsigned long number_t;

__device__ bool check_prime(number_t n)
{
  if (n <= 1)
		return false;
	if (n <= 3)
		return true;

	// This is checked so that we can skip
	// middle five numbers in below loop
	if (n % 2 == 0 || n % 3 == 0)
		return false;

	for (number_t i = 5; i*i <= n; i += 6)
		if(n % i == 0 || n % (i + 2) == 0)
			return false;

	return true;
}

__global__ void primes_in_range(number_t llimit, number_t ulimit, unsigned int *result)
{
	const number_t number = llimit + (blockIdx.x * blockDim.x) + threadIdx.x;
	if (number > ulimit)
	{
		return;
	}

  if (check_prime(number))
    atomicAdd(result, 1);
}


namespace Wrapper {
	void wrapper(number_t llimit, number_t ulimit)
	{

    unsigned int *result;
  	hipMallocManaged(&result, 4);
  	*result = 0;

    primes_in_range<<<10000, 1024>>>(llimit, ulimit, result);
  	hipDeviceSynchronize();
    printf("Primes found: %d\n", *result);

	}
}
