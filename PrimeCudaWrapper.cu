#include "hip/hip_runtime.h"
#include "PrimeCudaWrapper.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

typedef unsigned int number_t;

__device__ bool check_prime(number_t n) {
	if (n <= 1)
		return false;
	if (n <= 3)
		return true;

	// This is checked so that we can skip
	// middle five numbers in below loop
	if (n % 2 == 0 || n % 3 == 0)
		return false;

	for (number_t i = 5; i*i <= n; i += 6)
		if(n % i == 0 || n % (i + 2) == 0)
			return false;

	return true;
}

__global__ void primes_in_range(number_t a, number_t b, bool *primes) {
	const number_t number = a + (blockIdx.x * blockDim.x) + threadIdx.x;
	if (number > b)
		return;

	primes[number-a]=check_prime(number);

}


namespace CudaWrapper {
	std::list<number_t> cuda_wrapper(number_t a, number_t b) {

		int vec_size = b-a+1;
		thrust::host_vector<bool> primes_host(vec_size);
    thrust::fill(primes_host.begin(), primes_host.end(), false);
    thrust::device_vector<bool> primes = primes_host;

    bool* d_primes =  thrust::raw_pointer_cast(&primes[0]);

    primes_in_range<<<(b-a)/1000+1, 1024>>>(a, b, d_primes);
  	hipDeviceSynchronize();

    thrust::copy(primes.begin(), primes.end(), primes_host.begin());

    std::list<number_t> primes_list;

    for (int i = 0; i < vec_size; ++i)
      if (primes_host[i]){
        primes_list.push_back(a+i);
      }
    return primes_list;
	}
}
